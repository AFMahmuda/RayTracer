#include "Data3D.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <iostream>

Data3D& Data3D::operator+=(const Data3D& rhs) {

	hipError_t cudaStat; // hipMalloc status
	hipblasStatus_t stat; // CUBLAS functions status
	hipblasHandle_t handle; // CUBLAS context
	float *x, *y; // n- vector on the host
	x = v;
	y = rhs.v;
	float * d_x; // d_x - x on the device
	float * d_y; // d_y - y on the device
	hipMalloc((void **)& d_x, n * sizeof(*x)); // device
														   // memory alloc for x
	hipMalloc((void **)& d_y, n * sizeof(*y)); // device
														   // memory alloc for y
	stat = hipblasCreate(&handle); // initialize CUBLAS context
	stat = hipblasSetVector(n, sizeof(*v), x, 1, d_x, 1); // cp x- >d_x
	stat = hipblasSetVector(n, sizeof(*y), y, 1, d_y, 1); // cp y- >d_y
	float s = 1;
	stat = hipblasSaxpy(handle, n, &s, d_x, 1, d_y, 1); // d_y = s*d_x+d_y
	stat = hipblasGetVector(n, sizeof(float), d_y, 1, y, 1); // cp d_y - >y
	v = y; //this-> v  = y
	hipFree(d_x); // free device memory
	hipFree(d_y); // free device memory
	hipblasDestroy(handle); // destroy CUBLAS context

	return *this;
}

